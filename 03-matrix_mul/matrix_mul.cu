#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <cuda_device_runtime_api.h>

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

// 随机初始化
void ConstantInit(float *data, int size, float val) {
  for (int i = 0; i < size; ++i) {
    data[i] = rand() / (float)RAND_MAX;
  }
}

template <int BLOCK_SIZE>
__global__ void MatrixMulCUDA(float *C, float *A, float *B, int wA, int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv, int block_size, const dim3 &dimsA, const dim3 &dimsB) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A, hipHostMallocDefault));

  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B, hipHostMallocDefault));

  // Initialize host memory
  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C, hipHostMallocDefault));

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  checkCudaErrors(hipMalloc((void **)&d_A, mem_size_A));
  checkCudaErrors(hipMalloc((void **)&d_B, mem_size_B));
  checkCudaErrors(hipMalloc((void **)&d_C, mem_size_C));

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // copy host memory to device
  checkCudaErrors(
    hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
    hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

  // Setup execution parameters
  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

  // Create and start timer
  printf("Computing result using CUDA Kernel...\n");

  // Performs warmup operation using matrixMul CUDA kernel
  if (block_size == 16) {
    MatrixMulCUDA<16>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  } else {
    MatrixMulCUDA<32>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }

  printf("done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, stream));

  // Execute the kernel
  int nIter = 300;

  for (int j = 0; j < nIter; j++) {
    if (block_size == 16) {
      MatrixMulCUDA<16>
          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    } else {
      MatrixMulCUDA<32>
          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
  }

  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, stream));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance

  // Copy result from device to host
  checkCudaErrors(
    hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Checking computed result for correctness: ");
  bool correct = true;


  float eps = 1.e-4;
  for (int i = 0; i < dimsA.y; i++) {
    for (int j = 0; j < dimsB.x; j++) {
      float sum = 0.0;
      for (int k = 0; k < dimsA.x; k++) {
        sum += h_A[i * dimsA.x + k] * h_B[k * dimsB.x + j];
      }
      float c = h_C[i * dimsB.x + j];
      float abs_err = fabs(sum - c);
      if (abs_err > eps) {
        printf("Error!sum[%f],c[%f], abserr[%f]\n", sum, c, abs_err);
        correct = false;
        break;
      }
    }
    if (!correct) break;
  }

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  // Clean up memory
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  checkCudaErrors(hipStreamDestroy(stream));

  if (correct)
    return EXIT_SUCCESS;
  else
    return EXIT_FAILURE;
}

int main(int argc,char **argv)
{
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  int block_size = 32;

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);
}
